#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include "graph.h"
#include "bfsCPU.h"
#include <queue>

void bfsCPU(Graph &G) {
    for (int i = 0; i < G.rvector.size() - 1; i++) G.distances.push_back(-1);
    G.distances[G.root] = 0;
    std::queue<int> Q;
    Q.push(G.root);

    while (!Q.empty()) {
        int u = Q.front();
        Q.pop();

        for (int i = G.rvector[u]; i < G.rvector[u + 1]; i++) {
            int v = G.cvector[i];
            if (G.distances[v] == -1) {
                G.distances[v] = G.distances[u] + 1;
                Q.push(v);
            }
        }
    }
}

void runCpu(int startVertex, Graph &G) {
    G.root = startVertex;
    printf("Starting sequential bfs.\n");


    auto start = std::chrono::system_clock::now();
    bfsCPU(G);
    auto end = std::chrono::system_clock::now();


    float duration = 1000.0*std::chrono::duration<float>(end - start).count();
    for(int i = 0; i < G.distances.size(); i++) printf("%d ", G.distances[i]);

    printf("\n Elapsed time in milliseconds : %f ms.\n\n", duration);
    
}

int main(void)
{
    Graph G;
    G.cvector = {1, 3, 0, 2, 4, 4, 5, 7, 8, 6, 8};
    G.rvector = {0, 2, 5, 5, 6, 8, 9, 9, 11, 11};
    //run CPU sequential bfs
    runCpu(0, G);
    return 0;
}



extern "C" {

__global__
void simpleBfs(int N, int level, int *d_adjacencyList, int *d_edgesOffset,
               int *d_edgesSize, int *d_distance, int *d_parent, int *changed) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;
    int valueChange = 0;

    if (thid < N && d_distance[thid] == level) {
        int u = thid;
        for (int i = d_edgesOffset[u]; i < d_edgesOffset[u] + d_edgesSize[u]; i++) {
            int v = d_adjacencyList[i];
            if (level + 1 < d_distance[v]) {
                d_distance[v] = level + 1;
                d_parent[v] = i;
                valueChange = 1;
            }
        }
    }

    if (valueChange) {
        *changed = valueChange;
    }
}

__global__
void queueBfs(int level, int *d_adjacencyList, int *d_edgesOffset, int *d_edgesSize, int *d_distance, int *d_parent,
              int queueSize, int *nextQueueSize, int *d_currentQueue, int *d_nextQueue) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;

    if (thid < queueSize) {
        int u = d_currentQueue[thid];
        for (int i = d_edgesOffset[u]; i < d_edgesOffset[u] + d_edgesSize[u]; i++) {
            int v = d_adjacencyList[i];
            if (d_distance[v] == INT_MAX && atomicMin(&d_distance[v], level + 1) == INT_MAX) {
                d_parent[v] = i;
                int position = atomicAdd(nextQueueSize, 1);
                d_nextQueue[position] = v;
            }
        }
    }
}

//Scan bfs
__global__
void nextLayer(int level, int *d_adjacencyList, int *d_edgesOffset, int *d_edgesSize, int *d_distance, int *d_parent,
               int queueSize, int *d_currentQueue) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;

    if (thid < queueSize) {
        int u = d_currentQueue[thid];
        for (int i = d_edgesOffset[u]; i < d_edgesOffset[u] + d_edgesSize[u]; i++) {
            int v = d_adjacencyList[i];
            if (level + 1 < d_distance[v]) {
                d_distance[v] = level + 1;
                d_parent[v] = i;
            }
        }
    }
}

__global__
void countDegrees(int *d_adjacencyList, int *d_edgesOffset, int *d_edgesSize, int *d_parent,
                  int queueSize, int *d_currentQueue, int *d_degrees) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;

    if (thid < queueSize) {
        int u = d_currentQueue[thid];
        int degree = 0;
        for (int i = d_edgesOffset[u]; i < d_edgesOffset[u] + d_edgesSize[u]; i++) {
            int v = d_adjacencyList[i];
            if (d_parent[v] == i && v != u) {
                ++degree;
            }
        }
        d_degrees[thid] = degree;
    }
}

__global__
void scanDegrees(int size, int *d_degrees, int *incrDegrees) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;

    if (thid < size) {
        //write initial values to shared memory
        __shared__ int prefixSum[1024];
        int modulo = threadIdx.x;
        prefixSum[modulo] = d_degrees[thid];
        __syncthreads();

        //calculate scan on this block
        //go up
        for (int nodeSize = 2; nodeSize <= 1024; nodeSize <<= 1) {
            if ((modulo & (nodeSize - 1)) == 0) {
                if (thid + (nodeSize >> 1) < size) {
                    int nextPosition = modulo + (nodeSize >> 1);
                    prefixSum[modulo] += prefixSum[nextPosition];
                }
            }
            __syncthreads();
        }

        //write information for increment prefix sums
        if (modulo == 0) {
            int block = thid >> 10;
            incrDegrees[block + 1] = prefixSum[modulo];
        }

        //go down
        for (int nodeSize = 1024; nodeSize > 1; nodeSize >>= 1) {
            if ((modulo & (nodeSize - 1)) == 0) {
                if (thid + (nodeSize >> 1) < size) {
                    int next_position = modulo + (nodeSize >> 1);
                    int tmp = prefixSum[modulo];
                    prefixSum[modulo] -= prefixSum[next_position];
                    prefixSum[next_position] = tmp;

                }
            }
            __syncthreads();
        }
        d_degrees[thid] = prefixSum[modulo];
    }

}

__global__
void assignVerticesNextQueue(int *d_adjacencyList, int *d_edgesOffset, int *d_edgesSize, int *d_parent, int queueSize,
                             int *d_currentQueue, int *d_nextQueue, int *d_degrees, int *incrDegrees,
                             int nextQueueSize) {
    int thid = blockIdx.x * blockDim.x + threadIdx.x;

    if (thid < queueSize) {
        __shared__ int sharedIncrement;
        if (!threadIdx.x) {
            sharedIncrement = incrDegrees[thid >> 10];
        }
        __syncthreads();

        int sum = 0;
        if (threadIdx.x) {
            sum = d_degrees[thid - 1];
        }

        int u = d_currentQueue[thid];
        int counter = 0;
        for (int i = d_edgesOffset[u]; i < d_edgesOffset[u] + d_edgesSize[u]; i++) {
            int v = d_adjacencyList[i];
            if (d_parent[v] == i && v != u) {
                int nextQueuePlace = sharedIncrement + sum + counter;
                d_nextQueue[nextQueuePlace] = v;
                counter++;
            }
        }
    }
}

}

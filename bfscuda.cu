#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include "graph.h"
#include "bfscpu.h"
#include <queue>
#include <thrust/reduce.h>


void bfsCPU(Graph &G) {
    G.distances[G.root] = 0;
    std::queue<int> Q;
    Q.push(G.root);

    while (!Q.empty()) {
        int u = Q.front();
        Q.pop();

        for (int i = G.rvector[u]; i < G.rvector[u + 1]; i++) {
            int v = G.cvector[i];
            if (G.distances[v] == -1) {
                G.distances[v] = G.distances[u] + 1;
                Q.push(v);
            }
        }
    }
}

void runCpu(int startVertex, Graph &G) {
    G.root = startVertex;
    for (int i = 0; i < G.rvector.size() - 1; i++) G.distances.push_back(-1);
    printf("Starting sequential bfs.\n\n\n");
    auto start = std::chrono::system_clock::now();
    bfsCPU(G);
    auto end = std::chrono::system_clock::now();
    float duration = 1000.0*std::chrono::duration<float>(end - start).count();
    for(int i = 0; i < G.distances.size(); i++) printf("%d ", G.distances[i]);
    printf("\n \n\nElapsed time in milliseconds : %f ms.\n\n", duration);
    
}

__global__ void expansion(int* cvector, int* rvector, int* v_queue, int* e_queue, int *v_queuesize, int* e_queuesize, int* block_alloc_size, int* distances, int level, int extra)
{
    int tid = blockIdx.x *blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;
    __shared__ int prefixSum[1024];
    int u = v_queue[tid];
    int n = *v_queuesize;
    
    if(tid < extra) {

        if(*v_queuesize > 1024) {
            n = 1024;
        }
    }
    
        if(tid < extra && tid >= n) {
            prefixSum[tid] = 0;
        }
    

        if(tid < n) {
        //we create a block shared array of degrees of the elements of the current vertex frontier
        prefixSum[tid] = rvector[u + 1] - rvector[u];
        
        }
        
        if(tid < extra) {
        //1s of 3 scans in this algorithm - we calculate offsets for writing ALL neighbors into a block shared array
        // blelloch exclusive scan algorithm with upsweep to the left
        int offset = 1;
        for (int d = extra>>1; d > 0; d >>=1) {
            __syncthreads();
                    if(local_tid < d)
                    {
                    int ai = offset*(2*tid+1)-1;
                    int bi = offset*(2*tid+2)-1;
                    prefixSum[bi] += prefixSum[ai];
                    }
                    offset *= 2;
                
            
        }

        if (local_tid == 0) {
            int block = tid >> 10;
            // the efect of upsweep - reduction of the whole array (number of ALL neighbors)
            e_queuesize[0] = block_alloc_size[block + 1] = prefixSum[extra - 1];
            prefixSum[extra - 1] = 0;
            *v_queuesize = 0;

        }
        //downsweep - now our array prefixSum has become a prefix sum of numbers of neighbors
        for (int d = 1; d < extra; d *= 2) {
            offset >>= 1;
            __syncthreads();
            if (local_tid < d) {
                    int ai = offset*(2*tid+1)-1;
                    int bi = offset*(2*tid+2)-1;

                    int t = prefixSum[ai];
                    prefixSum[ai] = prefixSum[bi];
                    prefixSum[bi] += t;

            }
        }

        


        //scan on offsets produced by blocks in total
        if(gridDim.x > 1) {
            if(tid < gridDim.x) {
                for (int nodeSize = 2; nodeSize <= gridDim.x; nodeSize <<= 1) {
                    __syncthreads();
                    if ((tid & (nodeSize - 1)) == 0) {
                            int nextPosition = tid + (nodeSize >> 1);
                            block_alloc_size[tid] += block_alloc_size[nextPosition];
                        }
                    
                }
                if (tid == 0) {
                    *e_queuesize = block_alloc_size[tid];
                }
                for (int nodeSize = 1024; nodeSize > 1; nodeSize >>= 1) {
                    __syncthreads();
                    if ((tid & (nodeSize - 1)) == 0) {
                            int next_position = tid + (nodeSize >> 1);
                            int tmp = block_alloc_size[tid];
                            block_alloc_size[tid] -= block_alloc_size[next_position];
                            block_alloc_size[next_position] = tmp;
                        }
                    
                }
            }
        }
        int iter = 0;
        int temp = block_alloc_size[tid>>10];
        if (gridDim.x == 1) temp = 0;
        for(int i = rvector[u]; i < rvector[u + 1]; i++) {
            e_queue[iter + prefixSum[local_tid] + temp] = cvector[i];
            iter++;
        }

    }
}
__global__ void contraction(int* cvector, int* rvector, int* v_queue, int* e_queue, int *v_queuesize, int* e_queuesize, int* block_alloc_size, int* distances, int level, int extra)
{

    int tid = blockIdx.x *blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;
    extern __shared__ int array[];
    int* b1_initial = (int*)array; 
    int* b2_initial = b1_initial + *e_queuesize;
    int n = *e_queuesize;
    if(*e_queuesize > 1024) {
        n = 1024;
    }

    if(tid < extra && tid >= n) {
        b1_initial[tid] = 0;
    }

    if(local_tid < n) {
        b1_initial[local_tid] = 1;
        if(distances[e_queue[tid]] >= 0)
            b1_initial[local_tid] = 0;
    }

    if(tid < extra) {

        // we create a copy of this and make an array with scan of the booleans. this way we will know how many valid neighbors are there to check
        b2_initial[local_tid] = b1_initial[local_tid];

        int offset = 1;
        for (int d = extra>>1; d > 0; d >>=1) {
            __syncthreads();
                    if(local_tid < d)
                    {
                    int ai = offset*(2*tid+1)-1;
                    int bi = offset*(2*tid+2)-1;
                    b2_initial[bi] += b2_initial[ai];
                    }
                    offset *= 2;
                
            
        }

        if (local_tid == 0) {
            int block = tid >> 10;
            // the efect of upsweep - reduction of the whole array (number of ALL neighbors)
            v_queuesize[0] = block_alloc_size[block] = b2_initial[extra - 1];
            b2_initial[extra - 1] = 0;

        }
        //downsweep - now our array prefixSum has become a prefix sum of numbers of neighbors
        for (int d = 1; d < extra; d *= 2) {
            offset >>= 1;
            __syncthreads();
            if (local_tid < d) {
                    int ai = offset*(2*tid+1)-1;
                    int bi = offset*(2*tid+2)-1;

                    int t = b2_initial[ai];
                    b2_initial[ai] = b2_initial[bi];
                    b2_initial[bi] += t;

                
            }
        }
        __syncthreads();
        // now we have an array of neighbors, a mask signifying which we can copy further, and total number of elements to copy
    }

    //scan on offsets produced by blocks in total
    if(gridDim.x > 1) {
        if(tid < gridDim.x) {
            for (int nodeSize = 2; nodeSize <= gridDim.x; nodeSize <<= 1) {
                __syncthreads();
                if ((tid & (nodeSize - 1)) == 0) {
                    if (tid + (nodeSize >> 1) < gridDim.x) {
                        int nextPosition = tid + (nodeSize >> 1);
                        block_alloc_size[tid] += block_alloc_size[nextPosition];
                    }
                }
            }
            if (tid == 0) {
                *v_queuesize = block_alloc_size[tid];
            }
            for (int nodeSize = 1024; nodeSize > 1; nodeSize >>= 1) {
                __syncthreads();
                if ((tid & (nodeSize - 1)) == 0) {
                    if (tid + (nodeSize >> 1) < *v_queuesize) {
                        int next_position = tid + (nodeSize >> 1);
                        int tmp = block_alloc_size[tid];
                        block_alloc_size[tid] -= block_alloc_size[next_position];
                        block_alloc_size[next_position] = tmp;
                    }
                }
            }
        }
    }
    
    //now we compact
    if(b1_initial[local_tid])
    {
        int temp = block_alloc_size[tid>>10];
        if (gridDim.x == 1) temp = 0;
        distances[e_queue[local_tid]] = level + 1;
        v_queue[temp + b2_initial[local_tid]] = e_queue[local_tid];
    }
    }


void runGpu(int startVertex, Graph &G) {
    //declarations 
    G.root = startVertex;
    int level = 0;
    int num_blocks;
    int num_threads;
    int* v_queue;
    int* e_queue;
    int* block_alloc_size;
    int* distances;
    int* cvector;
    int* rvector;
    int *e_queuesize;
    int *v_queuesize;
    int num_vertices = G.rvector.size() - 1;

    //cuda unified memory allocations
    hipMallocManaged(&e_queuesize, sizeof(int));
    hipMallocManaged(&v_queuesize, sizeof(int));
    hipMallocManaged(&v_queue, num_vertices*sizeof(int));
    hipMallocManaged(&e_queue, num_vertices*sizeof(int));
    hipMallocManaged(&block_alloc_size, num_vertices*sizeof(int)/1024 + 1);
    hipMallocManaged(&distances, num_vertices*sizeof(int));
    
    //initializations 
    memset(distances, -1, num_vertices*sizeof(int));
    distances[G.root] = 0; 
    hipMallocManaged(&cvector, G.cvector.size()*sizeof(int));
    hipMallocManaged(&rvector, G.rvector.size()*sizeof(int));
    std::copy(G.cvector.begin(), G.cvector.end(), cvector);
    std::copy(G.rvector.begin(), G.rvector.end(), rvector);
    v_queue[0] = G.root;
    //for (int i = 0; i < G.rvector.size() - 1; i++) G.distances.push_back(-1);
    block_alloc_size[0] = 0;
    *v_queuesize = 1;
    level = 0;
    int mem;
    int extra;
    *e_queuesize = 0;
    printf("Starting cuda  bfs.\n\n\n");
    auto start = std::chrono::system_clock::now();
    while(*v_queuesize) { // it will work until the size of vertex frontier is 0
        extra = *v_queuesize;
        extra--;
        extra |= extra >> 1;
        extra |= extra >> 2;
        extra |= extra >> 4;
        extra |= extra >> 8;
        extra |= extra >> 16;
        extra++;
        //number of blocks scaled to the frontier size
        num_blocks = extra/1024 + 1;
        //if queue size is bigger than 1024 the numbers of threads has to be kept at 1024 because it's the maximum on CUDA
        if(num_blocks==1) num_threads = extra; else num_threads = 1024;
        //1st phase -> we expand vertex frontier into edge frontier by copying ALL possible neighbors
        //no threads stay idle apart from last block if num_threads > 1024, all SIMD lanes are utilized when reading from global memory

        expansion<<<num_blocks, num_threads>>>(cvector, rvector, v_queue, e_queue, v_queuesize, e_queuesize, block_alloc_size, distances, level, extra);
        hipDeviceSynchronize();
        extra = *e_queuesize;
        extra--;
        extra |= extra >> 1;
        extra |= extra >> 2;
        extra |= extra >> 4;
        extra |= extra >> 8;
        extra |= extra >> 16;
        extra++;
        //print newly produced edge frontier
        printf("E: size: %d, [", *e_queuesize); for(int i = 0; i < *e_queuesize; i++) printf("%d ", e_queue[i]); printf("]\n");
        num_blocks = (extra)/1024 + 1;
        if(num_blocks==1) num_threads = extra; else num_threads = 1024;
        mem = (extra)*2*sizeof(int);
        contraction<<<num_blocks, num_threads, mem>>>(cvector, rvector, v_queue, e_queue, v_queuesize, e_queuesize, block_alloc_size, distances, level, extra);
        hipDeviceSynchronize();
        printf("V: size: %d, [", *v_queuesize); for(int i = 0; i < *v_queuesize; i++) printf("%d ", v_queue[i]); printf("]\n");
        level++;
    }
    distances[G.root] = 0;
    auto end = std::chrono::system_clock::now();
    float duration = 1000.0*std::chrono::duration<float>(end - start).count();
    for(int i = 0; i < num_vertices; i++) printf("%d ", distances[i]);
    printf("\n \n\nElapsed time in milliseconds : %f ms.\n\n", duration);
    hipFree(v_queuesize);
    hipFree(e_queuesize);
    hipFree(v_queue);
    hipFree(e_queue);
    hipFree(block_alloc_size);
    hipFree(distances);
    hipFree(cvector);
    hipFree(rvector);
    
}


int main(void)
{
    Graph G;
    for(int i = 1; i < 993; i++){
        G.cvector.push_back(i);
    }
    for(int i = 0; i < 993; i++) {
        if(i < 32)
        G.rvector.push_back(i*31);
        else
        G.rvector.push_back(31*32);
    }
    //run CPU sequential bfs
    runCpu(0, G);
    //run GPU parallel bfs
    runGpu(0, G);
    return 0;
}
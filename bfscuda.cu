#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include "graph.h"
#include "bfscpu.h"
#include <queue>
#include <thrust/reduce.h>

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}

void bfsCPU(Graph &G) {
    G.distances[G.root] = 0;
    std::queue<int> Q;
    Q.push(G.root);

    while (!Q.empty()) {
        int u = Q.front();
        Q.pop();

        for (int i = G.rvector[u]; i < G.rvector[u + 1]; i++) {
            int v = G.cvector[i];
            if (G.distances[v] == -1) {
                G.distances[v] = G.distances[u] + 1;
                Q.push(v);
            }
        }
    }
}

void runCpu(int startVertex, Graph &G) {
    G.root = startVertex;
    for (int i = 0; i < G.rvector.size() - 1; i++) G.distances.push_back(-1);
    printf("Starting sequential bfs.\n\n\n");
    auto start = std::chrono::system_clock::now();
    bfsCPU(G);
    auto end = std::chrono::system_clock::now();
    float duration = 1000.0*std::chrono::duration<float>(end - start).count();
    for(int i = 0; i < G.distances.size(); i++) printf("%d ", G.distances[i]);
    printf("\n \n\nElapsed time in milliseconds : %f ms.\n\n", duration);
    
}

__global__ void expansion(int* cvector, int* rvector, int* v_queue, int* e_queue, int *v_queuesize, int* e_queuesize, int* v_block_alloc_size, int* e_block_alloc_size, int* distances, int level, int extra)
{
    int tid = blockIdx.x *blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;
    __shared__ int prefixSum[1024];
    int u = v_queue[tid];
    int n = *v_queuesize;
    int offset = 1;
    
    if(tid < extra) {
        if(*v_queuesize > 1024) {
            n = 1024;
        }
        else n = extra;
    }
    
    if(tid < extra && tid >= *v_queuesize) {
        prefixSum[local_tid] = 0;
    }


    if(tid < n) {
    //we create a block shared array of degrees of the elements of the current vertex frontier
        prefixSum[local_tid] = rvector[u + 1] - rvector[u];
    }
    
    if(tid < extra) {
    //1s of 4 scans in this algorithm - we calculate offsets for writing ALL neighbors into a block shared array
    // blelloch exclusive scan algorithm with upsweep to the left
        for (int d = n>>1; d > 0; d >>=1) {
            __syncthreads();
                    if(local_tid < d)
                    {
                    int ai = offset*(2*tid+1)-1;
                    int bi = offset*(2*tid+2)-1;
                    prefixSum[bi] += prefixSum[ai];
                    }
                    offset *= 2;
                
            
        }

        if (local_tid == 0) {
            int block = tid >> 10;
            // the efect of upsweep - reduction of the whole array (number of ALL neighbors)
            e_queuesize[0] = e_block_alloc_size[block] = prefixSum[n - 1];
            prefixSum[n - 1] = 0;
            *v_queuesize = 0;

        }
        //downsweep - now our array prefixSum has become a prefix sum of numbers of neighbors
        for (int d = 1; d < n; d *= 2) {
            offset >>= 1;
            __syncthreads();
            if (local_tid < d) {
                    int ai = offset*(2*tid+1)-1;
                    int bi = offset*(2*tid+2)-1;

                    int t = prefixSum[ai];
                    prefixSum[ai] = prefixSum[bi];
                    prefixSum[bi] += t;

            }
        }
}

    if(tid < gridDim.x && gridDim.x != 1) {
    //scan on offsets produced by blocks in 
        offset = 1;
        for (int d = gridDim.x>>1; d > 0; d >>=1) {
            __syncthreads();
                    if(local_tid < d)
                    {
                    int ai = offset*(2*tid+1)-1;
                    int bi = offset*(2*tid+2)-1;
                    e_block_alloc_size[bi] += e_block_alloc_size[ai];
                    }
                    offset *= 2;
                
            
        }

        if (tid == 0) {
        // the efect of upsweep - reduction of the whole array (number of ALL neighbors)
            e_queuesize[0] = e_block_alloc_size[n - 1];
            e_block_alloc_size[gridDim.x - 1] = 0;
            *v_queuesize = 0;

        }
        //downsweep - now our array prefixSum has become a prefix sum of numbers of neighbors
        for (int d = 1; d < gridDim.x; d *= 2) {
            offset >>= 1;
            __syncthreads();
            if (local_tid < d) {
                    int ai = offset*(2*tid+1)-1;
                    int bi = offset*(2*tid+2)-1;

                    int t = e_block_alloc_size[ai];
                    e_block_alloc_size[ai] = e_block_alloc_size[bi];
                    e_block_alloc_size[bi] += t;

            }
        }
}

    //saving into global edge frontier buffer
    int iter = 0;
    int temp = e_block_alloc_size[tid>>10];
    if (gridDim.x == 1) temp = 0;
    for(int i = rvector[u]; i < rvector[u + 1]; i++) {
        e_queue[iter + prefixSum[local_tid] + temp] = cvector[i];
        iter++;
    }

}

__global__ void contraction(int* cvector, int* rvector, int* v_queue, int* e_queue, int *v_queuesize, int* e_queuesize, int* v_block_alloc_size, int* e_block_alloc_size, int* distances, int level, int extra)
{
    int tid = blockIdx.x *blockDim.x + threadIdx.x;
    int local_tid = threadIdx.x;
    extern __shared__ int array[];
    int* b1_initial = (int*)array; 
    int n = *e_queuesize;
    int offset = 1;

    if(tid < extra) {
        if(*e_queuesize > 1024) {
            n = 1024;
        }
        else n = extra;
    }

    int* b2_initial = b1_initial + n*sizeof(int);


    if(tid < extra && tid >= *e_queuesize) {
        b1_initial[tid] = 0;
    }

    if(local_tid < n) {
        b1_initial[local_tid] = 1;
        if(distances[e_queue[tid]] >= 0)
            b1_initial[local_tid] = 0;
    }

    if(tid < extra) {
    // we create a copy of this and make an array with scan of the booleans. this way we will know how many valid neighbors are there to check
        b2_initial[local_tid] = b1_initial[local_tid];

        offset = 1;
        for (int d = n>>1; d > 0; d >>=1) {
            __syncthreads();
                    if(local_tid < d)
                    {
                    int ai = offset*(2*tid+1)-1;
                    int bi = offset*(2*tid+2)-1;
                    b2_initial[bi] += b2_initial[ai];
                    }
                    offset *= 2;
                
            
        }

        if (local_tid == 0) {
            int block = tid >> 10;
            // the efect of upsweep - reduction of the whole array (number of ALL neighbors)
            v_queuesize[0] = v_block_alloc_size[block] = b2_initial[n - 1];
            //printf("\n i, thread no %d, im setting index %d of block_offsets to %d\n", tid, block, b2_initial[n - 1]);
            b2_initial[n - 1] = 0;

        }
        //downsweep - now our array prefixSum has become a prefix sum of numbers of neighbors
        for (int d = 1; d < n; d *= 2) {
            offset >>= 1;
            __syncthreads();
            if (local_tid < d) {
                    int ai = offset*(2*tid+1)-1;
                    int bi = offset*(2*tid+2)-1;

                    int t = b2_initial[ai];
                    b2_initial[ai] = b2_initial[bi];
                    b2_initial[bi] += t;

                
            }
        }
        __syncthreads();
        // now we have an array of neighbors, a mask signifying which we can copy further, and total number of elements to copy
    }

    if(tid < gridDim.x && gridDim.x != 1) {
    //scan on offsets produced by blocks in 
            offset = 1;
            for (int d = gridDim.x>>1; d > 0; d >>=1) {
                __syncthreads();
                        if(local_tid < d)
                        {
                        int ai = offset*(2*tid+1)-1;
                        int bi = offset*(2*tid+2)-1;
                        v_block_alloc_size[bi] += v_block_alloc_size[ai];
                        }
                        offset *= 2;
                    
                
            }
    
            if (tid == 0) {
            // the efect of upsweep - reduction of the whole array (number of ALL neighbors)
                v_queuesize[0] = v_block_alloc_size[n - 1];
                v_block_alloc_size[gridDim.x - 1] = 0;
                *e_queuesize = 0;
    
            }
            //downsweep - now our array prefixSum has become a prefix sum of numbers of neighbors
            for (int d = 1; d < gridDim.x; d *= 2) {
                offset >>= 1;
                __syncthreads();
                if (local_tid < d) {
                        int ai = offset*(2*tid+1)-1;
                        int bi = offset*(2*tid+2)-1;
                        int t = v_block_alloc_size[ai];
                        v_block_alloc_size[ai] = v_block_alloc_size[bi];
                        v_block_alloc_size[bi] += t;
    
                }
            }
    }
    
    //now we compact
    if(b1_initial[local_tid])
    {
        int ver = e_queue[tid];
        int temp = v_block_alloc_size[tid>>10];
        if (gridDim.x == 1) temp = 0;
        distances[ver] = level + 1;
        v_queue[temp + b2_initial[local_tid]] = ver;
    }
}


void runGpu(int startVertex, Graph &G) {
    //declarations 
    G.root = startVertex;
    int level = 0;
    int num_blocks;
    int num_threads;
    int* v_queue;
    int* e_queue;
    int* v_block_alloc_size;
    int* e_block_alloc_size;
    int* distances;
    int* cvector;
    int* rvector;
    int *e_queuesize;
    int *v_queuesize;
    int num_vertices = G.rvector.size() - 1;

    //cuda unified memory allocations
    hipMallocManaged(&e_queuesize, sizeof(int));
    hipMallocManaged(&v_queuesize, sizeof(int));
    hipMallocManaged(&v_queue, num_vertices*sizeof(int));
    hipMallocManaged(&e_queue, num_vertices*sizeof(int));
    hipMallocManaged(&v_block_alloc_size, num_vertices*sizeof(int)/1024 + 1);
    hipMallocManaged(&e_block_alloc_size, num_vertices*sizeof(int)/1024 + 1);
    hipMallocManaged(&distances, num_vertices*sizeof(int));
    
    //initializations 
    memset(distances, -1, num_vertices*sizeof(int));
    distances[G.root] = 0; 
    hipMallocManaged(&cvector, G.cvector.size()*sizeof(int));
    hipMallocManaged(&rvector, G.rvector.size()*sizeof(int));
    std::copy(G.cvector.begin(), G.cvector.end(), cvector);
    std::copy(G.rvector.begin(), G.rvector.end(), rvector);
    v_queue[0] = G.root;
    //for (int i = 0; i < G.rvector.size() - 1; i++) G.distances.push_back(-1);
    v_block_alloc_size[0] = 0;
    e_block_alloc_size[0] = 0;
    *v_queuesize = 1;
    level = 0;
    int mem;
    int extra;
    *e_queuesize = 0;
    printf("Starting cuda  bfs.\n\n\n");
    auto start = std::chrono::system_clock::now();
    while(*v_queuesize) { // it will work until the size of vertex frontier is 0
        extra = *v_queuesize;
        extra--;
        extra |= extra >> 1;
        extra |= extra >> 2;
        extra |= extra >> 4;
        extra |= extra >> 8;
        extra |= extra >> 16;
        extra++;
        //number of blocks scaled to the frontier size
        num_blocks = extra/1025 + 1;
        //if queue size is bigger than 1024 the numbers of threads has to be kept at 1024 because it's the maximum on CUDA
        if(num_blocks==1) num_threads = extra; else num_threads = 1024;
        //1st phase -> we expand vertex frontier into edge frontier by copying ALL possible neighbors
        //no threads stay idle apart from last block if num_threads > 1024, all SIMD lanes are utilized when reading from global memory
        expansion<<<num_blocks, num_threads>>>(cvector, rvector, v_queue, e_queue, v_queuesize, e_queuesize, v_block_alloc_size, e_block_alloc_size, distances, level, extra);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        printf("\n\n\n");for(int i = 0; i < num_blocks; i++) printf("%d ", e_block_alloc_size[i]);printf("\n\n\n");
        extra = *e_queuesize;
        extra--;
        extra |= extra >> 1;
        extra |= extra >> 2;
        extra |= extra >> 4;
        extra |= extra >> 8;
        extra |= extra >> 16;
        extra++;
        //print newly produced edge frontier
        //printf("E: size: %d, [", *e_queuesize); for(int i = 0; i < *e_queuesize; i++) printf("%d ", e_queue[i]); printf("]\n");
        num_blocks = (extra)/1025 + 1;
        if(num_blocks==1) num_threads = extra; else num_threads = 1024;
        mem = (num_threads)*2*sizeof(int);
        contraction<<<num_blocks, num_threads, mem>>>(cvector, rvector, v_queue, e_queue, v_queuesize, e_queuesize, v_block_alloc_size, e_block_alloc_size, distances, level, extra);
        gpuErrchk( hipPeekAtLastError() );
        gpuErrchk( hipDeviceSynchronize() );
        printf("\n\n\n");for(int i = 0; i < num_blocks; i++) printf("%d ", v_block_alloc_size[i]); printf("\n\n\n");
        //printf("V: size: %d, [", *v_queuesize); for(int i = 0; i < *v_queuesize; i++) printf("%d ", v_queue[i]); printf("]\n");
        level++;
    }
    auto end = std::chrono::system_clock::now();
    float duration = 1000.0*std::chrono::duration<float>(end - start).count();
    printf("\n \n\nElapsed time in milliseconds : %f ms.\n\n", duration);
    hipFree(v_queuesize);
    hipFree(e_queuesize);
    hipFree(v_queue);
    hipFree(e_queue);
    hipFree(e_block_alloc_size);
    hipFree(v_block_alloc_size);
    hipFree(distances);
    hipFree(cvector);
    hipFree(rvector);
    
}


int main(void)
{
    Graph G;
    for(int i = 1; i < 1 + 10000; i++){
        G.cvector.push_back(i);
    }
    for(int i = 0; i < 1 + 10000 + 1; i++) {
        if(i < 1)
        G.rvector.push_back(0);
        else
        G.rvector.push_back(10000);
    }
    //run CPU sequential bfs
    runCpu(0, G);
    //run GPU parallel bfs
    runGpu(0, G);
    return 0;
}